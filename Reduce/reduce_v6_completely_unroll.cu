#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__device__ void warpReduce(volatile float *cache, unsigned int tid)
{
    cache[tid] += cache[tid + 32];

    cache[tid] += cache[tid + 16];

    cache[tid] += cache[tid + 8];

    cache[tid] += cache[tid + 4];

    cache[tid] += cache[tid + 2];

    cache[tid] += cache[tid + 1];

}
__global__ void reduce_completely_unroll(float *d_input, float *d_output)
{
    int tid = threadIdx.x;
    __shared__ float shared[THREAD_PER_BLOCK];
    float *input_begin = d_input + blockDim.x * blockIdx.x * 2;
    shared[tid] = input_begin[tid] + input_begin[tid + blockDim.x];
    __syncthreads();

    if (THREAD_PER_BLOCK >= 512)
    {
        if (tid < 256)
            shared[tid] += shared[tid + 256];
        __syncthreads();
    }
    if (THREAD_PER_BLOCK >= 256)
    {
        if (tid < 128)
            shared[tid] += shared[tid + 128];
        __syncthreads();
    }
    if (THREAD_PER_BLOCK >= 64)
    {
        if (tid < 64)
            shared[tid] += shared[tid + 64];
        __syncthreads();
    }

    if (tid < 32)
    {
        warpReduce(shared, tid);
    }
    if (tid == 0)
        d_output[blockIdx.x] = shared[0];
}

bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 0.005)
            return false;
    }
    return true;
}

int main()
{

    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK / 2;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));
    float *result = (float *)malloc(block_num * sizeof(float));
    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }
    // cpu calc
    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < 2 * THREAD_PER_BLOCK; j++)
        {
            cur += input[i * 2 * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    for (int i = 0; i < 10; i++)
        reduce_completely_unroll<<<Grid, Block>>>(d_input, d_output);
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, result, block_num))
        printf("the ans is right\n");
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
