#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__global__ void reduce_eliminate_warp_divergence(float *d_input, float *d_output)
{
    __shared__ float shared[THREAD_PER_BLOCK];
    float *input_begin = d_input + blockDim.x * blockIdx.x;
    shared[threadIdx.x] = input_begin[threadIdx.x];
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2)
    {
        if (threadIdx.x < blockDim.x / (2 * i))
        {
            int index = threadIdx.x * 2 * i;
            shared[index] += shared[index + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        d_output[blockIdx.x] = shared[0];
}

bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 0.005)
            return false;
    }
    return true;
}

int main()
{

    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK;
    float *output = (float *)malloc((N / THREAD_PER_BLOCK) * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, (N / THREAD_PER_BLOCK) * sizeof(float));
    float *result = (float *)malloc((N / THREAD_PER_BLOCK) * sizeof(float));
    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }
    // cpu calc
    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < THREAD_PER_BLOCK; j++)
        {
            cur += input[i * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(N / THREAD_PER_BLOCK, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);
    for (int i = 0; i < 10; i++)
        reduce_eliminate_warp_divergence<<<Grid, Block>>>(d_input, d_output);
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    if (check(output, result, block_num))
        printf("the ans is right\n");
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}